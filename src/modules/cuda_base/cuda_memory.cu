#include <stdio.h>
#include "cuda_common.h"
void allocate_device_data(void **buffer, long int number_of_bytes)
{
  check_cuda_error(hipMalloc(buffer, number_of_bytes));
}

extern "C" int cuda_update_device(void *dst, void *src, size_t nbytes)
{
  hipError_t error;

  error = hipMemcpy(dst, src, nbytes, hipMemcpyHostToDevice);
  check_cuda_error(error);

  return error;
}

extern "C" int cuda_update_host(void *dst, void *src, size_t nbytes)
{
  hipError_t error;

  error = hipMemcpy(dst, src, nbytes, hipMemcpyDeviceToHost);
  check_cuda_error(error);

  return error;
}

extern "C" int cuda_update_device_async(void *dst, void *src, size_t nbytes)
{
  hipError_t error;

  error = hipMemcpyAsync(dst, src, nbytes, hipMemcpyHostToDevice, 0);
  check_cuda_error(error);

  return error;
}

extern "C" int cuda_update_host_async(void *dst, void *src, size_t nbytes)
{
  hipError_t error;

  error = hipMemcpyAsync(dst, src, nbytes, hipMemcpyDeviceToHost, 0);
  check_cuda_error(error);

  return error;
}

extern "C" int cuda_device2device(void *dst, void *src, size_t nbytes)
{
  hipError_t error;

  error = hipMemcpy(dst, src, nbytes, hipMemcpyDeviceToDevice);
  check_cuda_error(error);

  return error;
}

extern "C" int cuda_update_device_async_with_stream(void *dst, void *src, size_t nbytes, hipStream_t *stream)
{
  hipError_t error;

  if ( stream )
  {
    error = hipMemcpyAsync(dst, src, nbytes, hipMemcpyHostToDevice, *stream);
  } else
  {
    error = hipMemcpyAsync(dst, src, nbytes, hipMemcpyHostToDevice, 0);
  }
  check_cuda_error(error);

  return error;
}

extern "C" int cuda_update_host_async_with_stream(void *dst, void *src, size_t nbytes, hipStream_t *stream)
{
  hipError_t error;

  if ( stream )
  {
    error = hipMemcpyAsync(dst, src, nbytes, hipMemcpyDeviceToHost, *stream);
  } else
  {
    error = hipMemcpyAsync(dst, src, nbytes, hipMemcpyDeviceToHost, 0);
  }
  check_cuda_error(error);

  return error;
}

extern "C" int cuda_device2device_with_stream(void *dst, void *src, size_t nbytes, hipStream_t *stream)
{
  hipError_t error;

  if ( stream )
  {
    error = hipMemcpyAsync(dst, src, nbytes, hipMemcpyDeviceToDevice, *stream);
  } else
  {
    error = hipMemcpyAsync(dst, src, nbytes, hipMemcpyDeviceToDevice, 0);
  }
  check_cuda_error(error);

  return error;
}

extern "C" int check_cuda_peek_at_last_error()
{
  hipError_t error;

  error = hipPeekAtLastError();

  check_cuda_error(error);

  return error;
}

extern "C" void cuda_error_printer(hipError_t code)
{
  fprintf(stderr,"hipError_t:%d: %s\n", code, hipGetErrorString(code));
}

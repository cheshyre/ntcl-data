#include <cuda_common.h>

extern "C" hipEvent_t * cuda_create_event()
{
        hipEvent_t *event;

        event = (hipEvent_t *) malloc(sizeof(hipEvent_t));
        check_cuda_error(hipEventCreateWithFlags(event, hipEventDisableTiming));
        return event;
}

extern "C" void cuda_destroy_event(hipEvent_t *event)
{
        check_cuda_error(hipEventDestroy(*event));
        free(event);
}

extern "C" hipError_t cuda_query_event(hipEvent_t *event)
{
        return hipEventQuery(*event);
}

extern "C" void cuda_record_event(hipEvent_t *event, hipStream_t *stream)
{
        check_cuda_error(hipEventRecord(*event, *stream));
}

extern "C" void cuda_record_event_without_stream(hipEvent_t *event)
{
        check_cuda_error(hipEventRecord(*event));
}

extern "C" hipStream_t * cuda_create_stream()
{
        hipStream_t *stream;

        stream = (hipStream_t *) malloc(sizeof(hipStream_t));
        check_cuda_error(hipStreamCreate(stream));
        return stream;
}

extern "C" void cuda_destroy_stream(hipStream_t *stream)
{
        check_cuda_error(hipStreamDestroy(*stream));
        free(stream);
}

extern "C" void cuda_stream_synchronize(hipStream_t *stream) {
        check_cuda_error(hipStreamSynchronize(*stream));
}

extern "C" hipError_t cuda_query_stream(hipStream_t *stream)
{
        return hipStreamQuery(*stream);
}

extern "C" void cuda_stream_wait_for_event(hipStream_t *stream, hipEvent_t *event)
{
        check_cuda_error(hipStreamWaitEvent(*stream, *event, 0));
}

extern "C" void cuda_event_synchronize(hipEvent_t *event)
{
    check_cuda_error(hipEventSynchronize(*event));
}

extern "C" void cuda_synchronize_all(void)
{
    check_cuda_error(hipDeviceSynchronize());
}
